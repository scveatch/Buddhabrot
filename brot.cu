
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#define GAIN 5
#define CUTOFF 1

// Declare Complex Structure:
struct Complex {
    double real;
    double imag;
};

// Device function to add two complex numbers
__device__ struct Complex addComplex(struct Complex a, struct Complex b) {
    struct Complex result;
    result.real = a.real + b.real;
    result.imag = a.imag + b.imag;
    return result;
}

// Create base on the GPU
__global__ void create_base(unsigned char *d_base, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = (i * size + j) * 3;

    if (i < size && j < size) {
        d_base[idx] = 0; // Red
        d_base[idx + 1] = 0; // Green
        d_base[idx + 2] = 0; // Blue
    }
}

// Device function to compute z_n^2 + c
__device__ struct Complex m_seq(struct Complex z_n, struct Complex c){
    struct Complex result;
    // Corrected the calculation to compute z_n^2
    result.real = (z_n.real * z_n.real) - (z_n.imag * z_n.imag);
    result.imag = (2 * (z_n.real * z_n.imag));
    struct Complex x = addComplex(result, c);
    // printf("%f + %fi\n", x.real, x.imag); // Confirm matches m_seq in brot.c
    return x;
}

// Complex to base; min and max come predefined in cuda. 
__device__ void c2b(struct Complex c, int size, int *x, int *y){
    *x = (int) ((c.real + 2) * size) / 4; 
    *y = (int) ((c.imag + 2) * size) / 4;
    *x = min(*x, size - 1);
    *y = min(*y, size - 1);
    *x = max(*x, 0);
    *y = max(*y, 0);
    return;
}

// Base to complex -- create complex "out", assign real and
// imaginary components and return complex out.
__device__ struct Complex b2c(int size, int x, int y){
    struct Complex out;
    out.real =  x * 4.0 / size - 2.0;
	out.imag =  y * 4.0 / size - 2.0;
    return out;
}

// Determine if a complex value c escapes within iters
// iterations.
__device__ int escapes(struct Complex c, int iters){
    struct Complex z_n = c; 
    for(int i = 0; i < iters; i++){
        z_n = m_seq(z_n, c);
        if(sqrt(z_n.real * z_n.real + z_n.imag * z_n.imag) > 2){
            return 1;
        }
    }
    return 0;
}

__device__ void one_val(unsigned char *d_base, int size, int iters, int color, struct Complex c) {
    struct Complex z_n = c;
    int x, y;

    // Check if value escapes within iters
    if (escapes(c, iters) == 0) {
        return;
    }
    
    for (int i = 0; i < iters; i++) {
        // Escape condition
        if (sqrt(z_n.real * z_n.real + z_n.imag * z_n.imag) > 2) {
            return;
        }
        
        c2b(z_n, size, &x, &y);
        x = (x < size - 1) ? x : size - 1;
        y = (y < size - 1) ? y : size - 1;
        
        int idx = ((x * size) + y) * 3;
        int v = d_base[idx + color] + 15;
        d_base[idx + color] = (v > 255) ? 255 : v;
        
        z_n = m_seq(z_n, c);
    }
}

__global__ void get_colors(unsigned char *d_base, int size, int iters) {
    int ilist[3] = {iters * 100, iters * 10, iters};

    // Iterate and run one_val
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
        for (int j = blockIdx.y * blockDim.y + threadIdx.y; j < size; j += blockDim.y * gridDim.y) {
            for (int color = 0; color < 3; color++) {
                struct Complex c = b2c(size, i, j);
                one_val(d_base, size, ilist[color], color, c);
            }
        }
    }
}

// Sigmoid correction
__device__ unsigned char sigmoid_correction(unsigned char pixel, double gain, double cut){
    double scaled = (double) pixel / 255.0;
    double corrected = 1.0 / (1.0 + exp(gain * (cut - scaled)));
    return (unsigned char) (corrected * 255);
}

// Run sigmoid on all pixels
__global__ void equalize(unsigned char *d_base, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < size && j < size) {
        for (int k = 0; k < 3; k++) {
            int idx = ((i * size) + j) * 3 + k;
            d_base[idx] = sigmoid_correction(d_base[idx], GAIN, CUTOFF);
        }
    }
}

// Function to write the output to a PPM file
void write_ppm(unsigned char *base, int size) {
    FILE *fp = fopen("cudabrot.ppm", "wb");
    fprintf(fp, "P6\n%d %d\n255\n", size, size);
    fwrite(base, sizeof(unsigned char), size * size * 3, fp);
    fclose(fp);
}

void make_brot(int size, int iters){
    unsigned char *d_base;
    hipMalloc((void **)&d_base, size * size * 3 * sizeof(unsigned char));

    dim3 blockSize(32, 32);
    dim3 gridSize((size + blockSize.x - 1) / blockSize.x, (size + blockSize.y - 1) / blockSize.y);

    // Initialize base on GPU
    create_base<<<gridSize, blockSize>>>(d_base, size);
    hipDeviceSynchronize();

    // Run kernel to compute Buddhabrot set
    get_colors<<<gridSize, blockSize>>>(d_base, size, iters);

    // Perform sigmoid correction
    equalize<<<gridSize, blockSize>>>(d_base, size);

    // Copy data from device to host
    unsigned char *base = (unsigned char *)malloc(size * size * 3 * sizeof(unsigned char));
    hipMemcpy(base, d_base, size * size * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Write base to ppm
    write_ppm(base, size);

    // Free everything
    hipFree(d_base);
    free(base);
}

int main(){
    // Launch kernel
    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    make_brot(15000, 10000);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Elapsed Time: %.3f seconds\n", elapsedTime / 1000);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}